//Class:        Introduction to Operating Systems
//Date:         11/16/18

// Base code by Bidur Bohara (LSU) in collaboration with Brygg Ullmer

// Compilation command: nvcc main.cu -o convert


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

//Declaration of class
class Bitmap
{
public:
    Bitmap();

    unsigned int bmpWidth;
    unsigned int bmpHeight;
    unsigned int bmpSize;

    unsigned char* readGrayBitmap(const char*file);
    void writeGrayBmp(unsigned char* data);

private:
    unsigned char* bmpHeader;
    unsigned int offset;
    unsigned int bitsPerPixel;

};

Bitmap::Bitmap()
{
    bmpWidth = 0;
    bmpHeight = 0;
    bmpSize = 0;
    offset = 0;
    bitsPerPixel = 0;
}

unsigned char* Bitmap::readGrayBitmap(const char *file)
{
    // Open bitmap file to read
    FILE *fp = fopen(file, "rb");
    if(!fp)
    {
        printf("Error! Cannot open input file.\n");
        return 0;
    }

    unsigned int status = 0;
    status = fseek(fp, 10, SEEK_SET); // Seek to width
    status = fread((void*)&offset, sizeof(unsigned int), 1, fp);

    status = fseek(fp, 18, SEEK_SET); // Seek to width
    status = fread((void*)&bmpWidth, sizeof(int), 1, fp);

    status = fseek(fp, 22, SEEK_SET); // Seek to height
    status = fread((void*)&bmpHeight, sizeof(int), 1, fp);

    status = fseek(fp, 28, SEEK_SET); // Seek to bits per pixel
    status = fread((void*)&bitsPerPixel, sizeof(unsigned short), 1, fp);

    status = fseek(fp, 34, SEEK_SET); // Seek to bitmap image size
    status = fread((void*)&bmpSize, sizeof(unsigned int), 1, fp);

    /// Read the Bitmap Header info.
    bmpHeader = new unsigned char[offset];
    status = fseek(fp, 0, SEEK_SET);
    status = fread((void*)bmpHeader, sizeof(unsigned char), offset, fp);

    /// Read the Bitmap image data.
    unsigned char* bmpData = new unsigned char[bmpSize];

    /// Seek to the position of image data.
    status = fseek(fp, offset, SEEK_SET);
    status = fread(bmpData, sizeof(unsigned char), bmpSize, fp);
    //bmpSize = status > 0 ? status : bmpSize;

    if(status){}

    fclose(fp);
    return bmpData;
}

void Bitmap::writeGrayBmp(unsigned char* data)
{
    FILE *wp = fopen("1.bmp", "wb");

    if(!data)
        printf("No data to be written!!!");

    unsigned int status = 0;

    status = fwrite((const void*)bmpHeader, sizeof(unsigned char),
                    offset, wp);
    status = fwrite((const void*)data, sizeof(unsigned char), bmpSize, wp);

    if(status){}

    fclose(wp);
}

/// Function that implements broken Sobel operator.
/// Returns image data after applying Sobel operator to the original image. Modified to function on a GPU.

__global__ void findEdge(const unsigned int w,
         const unsigned int h, const int threads, unsigned char* inData, unsigned char* image_sobeled)
{
    int gradient_X = 0;
    int gradient_Y = 0;
    int value = 0;

    int sobel_x[3][3] = { { 1, 0,-1},
                        { 2, 0,-2},
                        { 1, 0,-1}};

    int sobel_y[3][3] = { { 1, 2, 1},
                        { 0, 0, 0},
                        {-1,-2,-1}};

    int chunksize = ceilf((float)(h/threads));

    // The FOR loop apply Sobel operator
    // to bitmap image data in per-pixel level.
    for(unsigned int y = blockIdx.x*chunksize+1; y < ((blockIdx.x+1)*chunksize)+1; ++y)
    {
        if(y>=h)
        {

        }
        else
        {
          for(unsigned int x = 1; x < w-1; ++x)
          {
              // Compute gradient in +ve x direction
              gradient_X = sobel_x[0][0] * inData[ (x-1) + (y-1) * w ]
                      + sobel_x[0][1] * inData[  x    + (y-1) * w ]
                      + sobel_x[0][2] * inData[ (x+1) + (y-1) * w ]
                      + sobel_x[1][0] * inData[ (x-1) +  y    * w ]
                      + sobel_x[1][1] * inData[  x    +  y    * w ]
                      + sobel_x[1][2] * inData[ (x+1) +  y    * w ]
                      + sobel_x[2][0] * inData[ (x-1) + (y+1) * w ]
                      + sobel_x[2][1] * inData[  x    + (y+1) * w ]
                      + sobel_x[2][2] * inData[ (x+1) + (y+1) * w ];

                      // Compute gradient in +ve y direction
                      gradient_Y = sobel_y[0][0] * inData[ (x-1) + (y-1) * w ]
                      + sobel_y[0][1] * inData[  x    + (y-1) * w ]
                      + sobel_y[0][2] * inData[ (x+1) + (y-1) * w ]
                      + sobel_y[1][0] * inData[ (x-1) +  y    * w ]
                      + sobel_y[1][1] * inData[  x    +  y    * w ]
                      + sobel_y[1][2] * inData[ (x+1) +  y    * w ]
                      + sobel_y[2][0] * inData[ (x-1) + (y+1) * w ]
                      + sobel_y[2][1] * inData[  x    + (y+1) * w ]
                      + sobel_y[2][2] * inData[ (x+1) + (y+1) * w ];

                      value = (int)ceilf((sqrtf((float)(gradient_X * gradient_X + gradient_Y * gradient_Y))));

                      if(value>255) value=255;
                      if(value<0) value=0;

                      image_sobeled[ x + y*w ] = 255 - value;
          }
        }
    }
    // Thanks to Thomas Peters.
}

// Creates and runs a specified number of threads inside main function. Workload is divided amongst every GPU thread
int main(int argc, char *argv[])
{
    char* bmpFile; //Name of file to convert
    char* threads;
    int threadCount;

    /// Memory to hold input image data
    unsigned char* inData;
    unsigned char* image_sobeled;

    if( argc < 3) //./convert [FILENAME] [THREADS]
      {
	       printf("Filename and thread count arguments required!\n");
         printf("Usage: ./convert [FILENAME] [THREADS]\n");
	       return 0;
      }
    else
      bmpFile = argv[1]; //Save filename to pointer
      threads = argv[2]; //Save # of threads to spin up

    threadCount = atoi(threads);

    if(threadCount<=0)
    {
      printf("Thread count cannot be 0 or negative!\n");
      return 0;
    }

    /// Open and read bmp file.
    Bitmap *image = new Bitmap(); //Initialize Bitmap object
    unsigned char*data = image->readGrayBitmap(bmpFile); //Use member method to read

    //Allocate CUDA Unified Memory to allow CPU and GPU to access same memory space
    hipMallocManaged(&image_sobeled, image->bmpSize*sizeof(unsigned char));
    hipMallocManaged(&inData, image->bmpSize*sizeof(unsigned char));

    //Allocation done, initialize
    for(int i = 0; i < image->bmpSize; i++)
    {
      image_sobeled[i] = 255; //Refactoring vector initialization
    }

    for(int n = 0; n < image->bmpSize; n++)
    {
      inData[n] = data[n];
    }

    //Execute CUDA kernel for specified number of threads
    findEdge<<<threadCount, 1>>>(image->bmpWidth, image->bmpHeight, threadCount, inData, image_sobeled);//Apply Sobel

    //Check for any errors thrown by the kernel
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
    printf("Error: %s\n", hipGetErrorString(err));
    }

    hipDeviceSynchronize();//Wait for GPU's to finish processing

    printf("Threads done!\n");

    /// Write image data passed as argument to a bitmap file
    image->writeGrayBmp(&image_sobeled[0]);

    //Clean up
    hipFree(image_sobeled);
    hipFree(inData);
    delete data;

    return 0;
}